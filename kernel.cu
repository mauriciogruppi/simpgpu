#include "hip/hip_runtime.h"
#include "kernel.cuh"
#include <iostream>

//get position from header array
//Size of each structure on arrays
#define HEADER_SIZE  3
#define FACE_SIZE 3
#define VERTEX_SIZE 3
#define FACE_DATA_BATCH_SIZE 50
#define EDGE_DATA_BATCH_SIZE 50
#define QUADRIC_SIZE 16 //Quadric for a vertex is a 4x4 matrix
#define EDGE_SIZE 2

//UGRID
#define CELL_HEADER_SIZE 2

//ACCESS Vertex
#define getFaceVertexId(face,vertex) faces[FACE_SIZE*face+vertex]
#define getFaceHeaderPos(vertex) vert_face_header[HEADER_SIZE*vertex]
#define getFaceCurrSize(vertex) vert_face_header[HEADER_SIZE*vertex+1]
#define faceIncreaseSize(vertex) vert_face_header[HEADER_SIZE*vertex+1]++
#define faceDecreaseSize(vertex) vert_face_header[HEADER_SIZE*vertex+1]--
#define getFaceId(vid,p) vert_face_data[getFaceHeaderPos(vid)+p]
#define getX(vid) vertices[VERTEX_SIZE*vid]
#define getY(vid) vertices[VERTEX_SIZE*vid+1]
#define getZ(vid) vertices[VERTEX_SIZE*vid+2]

//ACCESS TO EDGES
#define getEdgeVertexId(edge,vid) edges[EDGE_SIZE*edge+vid] //Get vertex id (0 or 1) of edge
#define getEdgeFromHeaderPos(vid) vert_edge_from_header[HEADER_SIZE*vid]
#define getEdgeFromCurrSize(vid) vert_edge_from_header[HEADER_SIZE*vid+1]
#define edgeFromIncreaseSize(vid) assert(vert_edge_from_header[HEADER_SIZE*vid+1] < EDGE_DATA_BATCH_SIZE); vert_edge_from_header[HEADER_SIZE*vid+1]++
#define edgeFromDecreaseSize(vid) vert_edge_from_header[HEADER_SIZE*vid+1]--
#define getEdgeFromDataId(vid,p) vert_edge_from_data[getEdgeFromHeaderPos(vid)+p]

//edge_to
#define edgeToIncreaseSize(vid) assert(vert_edge_from_header[HEADER_SIZE*vid+1] < EDGE_DATA_BATCH_SIZE); vert_edge_to_header[HEADER_SIZE*vid+1]++
#define edgeToDecreaseSize(vid) vert_edge_to_header[HEADER_SIZE*vid+1]--
#define getEdgeToHeaderPos(vid) vert_edge_to_header[HEADER_SIZE*vid]
#define getEdgeToCurrSize(vid) vert_edge_to_header[HEADER_SIZE*vid+1]
#define getEdgeToDataId(vid,p) vert_edge_to_data[getEdgeToHeaderPos(vid)+p]

//Access
#define getPlacementX(vid1,vid2) (getX(vid1)+getX(vid2))/2
#define getPlacementY(vid1,vid2) (getY(vid1)+getY(vid2))/2
#define getPlacementZ(vid1,vid2) (getZ(vid1)+getZ(vid2))/2

//UNIFORM GRID
#define getCellHeaderPos(cell) cell_header[CELL_HEADER_SIZE*cell]
#define getCellHeaderSize(cell) cell_header[CELL_HEADER_SIZE*cell+1]
#define increaseCellSize(cell) cell_header[CELL_HEADER_SIZE*cell+1]++
#define getCellVertexId(cell,p) cell_vertices[cell*n_vertices+p]
#define HEAP_HEADER_SIZE 3
#define HEAP_SIZE_VERTEX_RATIO 8
#define getHeapHead(cell) cell_heap_header[HEAP_HEADER_SIZE*(cell)]
#define getCellHeapSize(cell) cell_heap_header[HEAP_HEADER_SIZE*(cell)+1]
#define getCellMaxHeapSize(cell) cell_heap_header[HEAP_HEADER_SIZE*(cell)+2]


//======DEVICE (GLOBAL) VARIABLES======
//VERTICES
__device__ double* vertices;
__device__ bool* vertex_removed;
__device__ double* quadrics;

//FACES
__device__ int* faces;
__device__ int* vert_face_header;
__device__ int* vert_face_data;
__device__ bool* face_removed;

//EDGES
__device__ int* edges;
__device__ double* edge_cost;
__device__ int* edge_queue;
__device__ int* vert_edge_from_header;
__device__ int* vert_edge_from_data;
__device__ int* vert_edge_to_header;
__device__ int* vert_edge_to_data;
__device__ bool* edge_removed;

//UNIFORM GRID
__device__ int n_cells;

//VERTEX
__device__ int* cell_data;
__device__ int* cell_header;
__device__ int* vertex_in_cell;
__device__ int* initial_vertices;
__device__ int* cell_vertices;
__device__ int* cell_vertices_size;

//EDGE
__device__ int* cell_heap;
__device__ int* cell_heap_header;
__device__ elem* cell_heap_data;
__device__ int* cell_heap_size;


//UNIFORM GRID DIM AND OFFSET
__device__ double dim[3];
__device__ double offset[3];
__device__ int grid_res;
__device__ double* bbox;


//NO.
__device__ int n_edges;
__device__ int n_faces;
__device__ int n_vertices;

//=====



//=====HEAP FUNCTIONS ======

__device__ bool d_compare(const elem &left,const elem &right)
{
  return left.cost < right.cost;
}

//=====HEAP FUNCTIONS ======

//Move node up
__device__ void d_percUP(int i, elem* heap)
{
  while(i/2 > 0)
  {
    if(d_compare(heap[i], heap[i/2]))
    {
      elem aux = heap[i];
      heap[i] = heap[i/2];
      heap[i/2] = aux;
    }
    i/=2;
  }
}

//d_insert element e into heap of size n
__device__ void d_insert(elem e, elem* heap, int& n)
{
  // cout << "Elem id: " << e.id << endl;
  heap[n+1] = e;
  n++;
  d_percUP(n, heap);

}

//Find child of least cost
__device__ int d_minChild(int i, elem* heap, int n)
{
  //Does not have right child
  if (i * 2 + 1 > n) return i*2;
  else{
    if(d_compare(heap[2*i],heap[2*i+1])) return 2*i;
    else return 2*i+1;
  }
}

//Move node down the heap
__device__ void d_percDown(int i, elem* heap, int n)
{
  while(i*2 <= n)
  {
    int mc = d_minChild(i, heap, n);
    if(d_compare(heap[mc],heap[i]))
    {
      elem tmp = heap[mc];
      heap[mc] = heap[i];
      heap[i] = tmp;
    }
     i = mc;
  }
}

//Pop element of least value
__device__ elem d_pop(elem* heap, int& n)
{
  elem ret = heap[1];
  heap[1] = heap[n];
  n--;
  d_percDown(1, heap, n);
  return ret;
}

//Turn vector into heap or update heap
__device__ void d_heapify(elem* vec, int n)
{
  int i = n/2;
  while( i > 0)
  {
    d_percDown(i,vec,n);
    i--;
  }
}


//===END OF HEAP===



__device__ bool d_isEntirelyInCell(int eid)
{
  //Return if both endpoints are withing the same cell
  return vertex_in_cell[getEdgeVertexId(eid,0)] == vertex_in_cell[getEdgeVertexId(eid,1)];
}

__device__ bool d_isCrownInCell(int vid)
{
  //Test edges leaving (from) vid to see if their endpoints also lie within vid's cell
  //printf("Vertex %d\n", vid);
  for(int i = 0 ; i < getEdgeFromCurrSize(vid); ++i)
  {
    //printf("From\n");
    //Check edge i from vid

  //  printf("edges from %d\n", getEdgeFromCurrSize(vid));
    int eid = vert_edge_from_data[EDGE_DATA_BATCH_SIZE*vid+i];
    //int eid = getEdgeFromDataId(vid,i);
    int endp = getEdgeVertexId(eid,1);

    if(vertex_in_cell[endp] != vertex_in_cell[vid])
      return false;
  }

  //Test edges arriving (to) vid
  for(int i = 0 ; i < getEdgeToCurrSize(vid); ++i)
  {
    //printf("To\n");
    //int eid = getEdgeToDataId(vid,i);
    int eid = vert_edge_to_data[EDGE_DATA_BATCH_SIZE*vid+i];
    int endp = getEdgeVertexId(eid,0);
    if(vertex_in_cell[endp] != vertex_in_cell[vid])
      return false;
  }
  return true;
}


__global__ void computeGridEdges()
{

  int i = threadIdx.x;

  printf("Thread %d\n", i);

  for(int j = 0; j < cell_vertices_size[i]; ++j)
  {
    //printf("j | cell_vertices_size %d | %d\n", j, cell_vertices_size[i]);
    for(int k = 0; k < getEdgeFromCurrSize(getCellVertexId(i,j)); ++k)
    {

      int vid = getCellVertexId(i,j);
      //int eid = getEdgeFromDataId(getCellVertexId(i,j),k);
      int eid = vert_edge_from_data[EDGE_DATA_BATCH_SIZE*vid+k];

      if(d_isEntirelyInCell(eid) && d_isCrownInCell(getEdgeVertexId(eid,0)) && d_isCrownInCell(getEdgeVertexId(eid,1)))
      {

        elem temp = {eid,edge_cost[eid]};
        //d_insert(temp, cell_heap_data+getHeapHead(i), getCellHeapSize(i));
      }

    }
  }



}

__global__ void initUniformGrid()
{

  printf("grid_res %d\n", grid_res);
  n_cells = grid_res*grid_res;
  n_cells = n_cells*grid_res;
  offset[0] = bbox[0];
  offset[1] = bbox[1];
  offset[2] = bbox[2];

  dim[0] = bbox[3]/grid_res;
  dim[1] = bbox[4]/grid_res;
  dim[2] = bbox[5]/grid_res;

  for(int i = 0; i < n_cells; ++i)
  {
    initial_vertices[i] = 0;
    cell_vertices_size[i] = 0;

    getHeapHead(i) = 0;
    getCellHeapSize(i) = 0;
  }

  for(int i = 0; i < n_vertices; ++i)
  {
    if(vertex_removed[i])
    {
      continue;
    }
    int cx = (getX(i) - offset[0])/dim[0];
    cx -= cx/grid_res;
    int cy = (getY(i) - offset[1])/dim[1];
    cy -= cy/grid_res;
    long long cz = (getZ(i) - offset[2])/dim[2];
    cz -= cz/grid_res;
    int cpos = cx + grid_res*cy + grid_res*grid_res*cz;

    vertex_in_cell[i] = cpos;
    initial_vertices[cpos]++;

    cell_vertices[cpos*n_vertices+cell_vertices_size[cpos]] = i;
    cell_vertices_size[cpos]++;
  }

  for(int i = 0; i < n_cells; ++i)
  {
    getCellHeapSize(i) = 0;
    getCellMaxHeapSize(i) = HEAP_SIZE_VERTEX_RATIO*initial_vertices[i];

    if(i!=0){
      getHeapHead(i) = getHeapHead(i-1) + getCellMaxHeapSize(i-1)+1;

    }
    else{
      getHeapHead(i) = 0;
    }
  }

  //Calculate edges for each cell separately
  computeGridEdges<<<1,n_cells>>>();

}


//Set pointers of global variables
__global__ void initDevice(environmentList)
{
  vertices = d_vertices;
  vertex_removed = d_vertex_removed;
  quadrics = d_quadrics;
  faces = d_faces;
  vert_face_header = d_vert_face_header;
  vert_face_data = d_vert_face_data;
  face_removed = d_face_removed;
  edges = d_edges;
  edge_cost = d_edge_cost;
  edge_queue = d_edge_queue;
  vert_edge_from_header = d_vert_edge_from_header;
  vert_edge_from_data = d_vert_edge_from_data;
  vert_edge_to_header = d_vert_edge_to_header;
  vert_edge_to_data = d_vert_edge_to_data;
  edge_removed = d_edge_removed;
  vertex_in_cell = d_vertex_in_cell;
  initial_vertices = d_initial_vertices;
  cell_vertices = d_cell_vertices;
  cell_vertices_size = d_cell_vertices_size;
  cell_heap_header = d_cell_heap_header;
  cell_heap_data = d_cell_heap_data;
  cell_heap_size = d_cell_heap_size;
  bbox = d_bbox;
  n_vertices = d_n_sizes[0];
  n_faces = d_n_sizes[1];
  n_edges = d_n_sizes[2];
  grid_res = d_n_sizes[3];
}

void initDeviceEnvironment(hostList,environmentReferenceList)
{

  //INTEGERS
  d_n_edges = h_n_edges;
  d_n_faces = h_n_faces;
  d_n_cells = h_n_cells;
  d_n_vertices = h_n_vertices;

  //INIT_DATA_STRUCTURES
  int size = FACE_SIZE*h_n_faces*sizeof(int);
  hipMalloc(&d_faces,size);
  hipMemcpy(d_faces, h_faces, size, hipMemcpyHostToDevice);

  size = h_n_faces*sizeof(bool);
  hipMalloc(&d_face_removed,size);
  hipMemcpy(d_face_removed, h_face_removed, size, hipMemcpyHostToDevice);

  size = VERTEX_SIZE*h_n_vertices*sizeof(double);
  hipMalloc(&d_vertices, size);
  hipMemcpy(d_vertices, h_vertices, size, hipMemcpyHostToDevice);

  size = h_n_vertices*sizeof(bool);
  hipMalloc(&d_vertex_removed, size);
  hipMemcpy(d_vertex_removed, h_vertex_removed, size, hipMemcpyHostToDevice);

  size = 16*h_n_vertices*sizeof(double);
  hipMalloc(&d_quadrics,size);
  hipMemcpy(d_quadrics, h_quadrics, size, hipMemcpyHostToDevice);

  size = HEADER_SIZE*h_n_vertices*sizeof(int);
  hipMalloc(&d_vert_face_header, size);
  hipMemcpy(d_vert_face_header, h_vert_face_header, size, hipMemcpyHostToDevice);

  size = FACE_DATA_BATCH_SIZE*h_n_vertices*sizeof(int);
  hipMalloc(&d_vert_face_data, size);
  hipMemcpy(d_vert_face_data, h_vert_face_data, size, hipMemcpyHostToDevice);

  //INIT_EDGES
  size = h_n_faces*6*sizeof(int);
  hipMalloc(&d_edges, size);
  hipMemcpy(d_edges, h_edges, size, hipMemcpyHostToDevice);

  size = HEADER_SIZE*h_n_vertices*sizeof(int);
  hipMalloc(&d_vert_edge_from_header, size);
  hipMemcpy(d_vert_edge_from_header, h_vert_edge_from_header, size, hipMemcpyHostToDevice);

  size = EDGE_DATA_BATCH_SIZE*h_n_vertices*sizeof(int);
  hipMalloc(&d_vert_edge_from_data, size);
  hipMemcpy(d_vert_edge_from_data, h_vert_edge_from_data, size, hipMemcpyHostToDevice);

  size = HEADER_SIZE*h_n_vertices*sizeof(int);
  hipMalloc(&d_vert_edge_to_header, size);
  hipMemcpy(d_vert_edge_to_header, h_vert_edge_to_header, size, hipMemcpyHostToDevice);

  size = EDGE_DATA_BATCH_SIZE*h_n_vertices*sizeof(int);
  hipMalloc(&d_vert_edge_to_data, size);
  hipMemcpy(d_vert_edge_to_data, h_vert_edge_to_data, size, hipMemcpyHostToDevice);

  size = h_n_edges*sizeof(double);
  hipMalloc(&d_edge_cost, size);
  hipMemcpy(d_edge_cost, h_edge_cost, size, hipMemcpyHostToDevice);

  size = h_n_edges*sizeof(bool);
  hipMalloc(&d_edge_removed, size);
  hipMemcpy(d_edge_removed, h_edge_removed, size, hipMemcpyHostToDevice);

  //UNIFORM_GRID
  size = h_n_vertices*sizeof(int);
  hipMalloc(&d_vertex_in_cell, size);
  hipMemcpy(d_vertex_in_cell, h_vertex_in_cell, size, hipMemcpyHostToDevice);

  size = h_n_cells*sizeof(int);
  hipMalloc(&d_initial_vertices, size);
  hipMemcpy(d_initial_vertices, h_initial_vertices, size, hipMemcpyHostToDevice);

  size = h_n_cells*sizeof(int);
  hipMalloc(&d_cell_vertices_size, size);
  hipMemcpy(d_cell_vertices_size, h_cell_vertices_size, size, hipMemcpyHostToDevice);

  size = h_n_cells*h_n_vertices*sizeof(int);
  hipMalloc(&d_cell_vertices, size);
  hipMemcpy(d_cell_vertices, h_cell_vertices, size, hipMemcpyHostToDevice);

  size = h_n_cells*(HEADER_SIZE)*sizeof(int);
  hipMalloc(&d_cell_heap_header, size);
  hipMemcpy(d_cell_heap_header, h_cell_heap_header, size, hipMemcpyHostToDevice);

  size = h_n_vertices*sizeof(elem)*HEAP_SIZE_VERTEX_RATIO;
  hipMalloc(&d_cell_heap_data, size);
  hipMemcpy(d_cell_heap_data, h_cell_heap_data, size, hipMemcpyHostToDevice);

  size = h_n_cells*sizeof(int);
  hipMalloc(&d_cell_heap_size, size);
  hipMemcpy(d_cell_heap_size, h_cell_heap_size, size, hipMemcpyHostToDevice);

  //bbox
  size = 6*sizeof(double);
  hipMalloc(&d_bbox, size);
  hipMemcpy(d_bbox, h_bbox, size, hipMemcpyHostToDevice);

  //sizes
  size = 4*sizeof(int);
  hipMalloc(&d_n_sizes, size);
  hipMemcpy(d_n_sizes, h_n_sizes, size, hipMemcpyHostToDevice);

  initDevice<<<1,1>>>(environmentArgumentList);
  hipDeviceSynchronize();
}

void pullFromDevice(hostList, environmentReferenceList)
{

  std::cerr << "Pulling from device...\n";

  int size = FACE_SIZE*h_n_faces*sizeof(int);
  hipMemcpy(h_faces, d_faces, size, hipMemcpyDeviceToHost);

  size = h_n_faces*sizeof(bool);
  hipMemcpy(h_face_removed, d_face_removed, size, hipMemcpyDeviceToHost);

  size = VERTEX_SIZE*h_n_vertices*sizeof(double);
  hipMemcpy(h_vertices, d_vertices, size, hipMemcpyDeviceToHost);

  size = h_n_vertices*sizeof(bool);
  hipMemcpy(h_vertex_removed, d_vertex_removed, size, hipMemcpyDeviceToHost);

  size = HEADER_SIZE*h_n_vertices*sizeof(int);
  hipMemcpy(h_vert_face_header, d_vert_face_header, size, hipMemcpyDeviceToHost);
}

void initializeUniformGrid()
{
  std::cerr << "Initializing uniform grid (device)...\n";
  initUniformGrid<<<1,1>>>();
  hipDeviceSynchronize();
}

void freeDevice(int* d_a)
{
  hipFree(d_a);
}
